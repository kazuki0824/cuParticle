#include "hip/hip_runtime.h"
/*
 * kernel.cu
 *
 *  Created on: 2018/04/21
 *      Author: kazuki
 */


#include <hiprand/hiprand_kernel.h>
#include "nd_noise.hcu"
#include "likelihood.hcu"

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel
// 6, we observe the maximum specified number of blocks, because each thread in
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{

    //get device capability, to avoid block/grid size exceed the upper bound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
        blocks = (n + threads - 1) / threads;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }

    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    if (whichKernel == 6)
    {
        blocks = MIN(maxBlocks, blocks);
    }
}


template <class T, unsigned int blockSize>
__global__ void
reduce5(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;

    if (i + blockSize < n)
        mySum += g_idata[i+blockSize];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks,
       int whichKernel, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    // choose which of the optimized versions of reduction to launch

    switch (threads)
    {
    case 512:
    	reduce5<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case 256:
    	reduce5<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case 128:
    	reduce5<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case 64:
    	reduce5<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case 32:
    	reduce5<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case 16:
    	reduce5<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case  8:
    	reduce5<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case  4:
    	reduce5<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case  2:
    	reduce5<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;

    case  1:
    	reduce5<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
    	break;
    }

}

// Instantiate the reduction function for 3 types
template void
reduce<int>(int size, int threads, int blocks,
            int whichKernel, int *d_idata, int *d_odata);

template void
reduce<float>(int size, int threads, int blocks,
              int whichKernel, float *d_idata, float *d_odata);

template void
reduce<double>(int size, int threads, int blocks,
               int whichKernel, double *d_idata, double *d_odata);



__device__ void pf(float3 current)
{




	float3 noise = getSystemNoise(&s);

	float3 prediction = suggest_distribution(current ,noise); //TODO:現在の状態"current"はどこから入手するか

	float l = likelihood(prediction); //予測のもっともらしさを計算する

	/*リダクションで尤度の総和を導出*/



}




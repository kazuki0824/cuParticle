#include "hip/hip_runtime.h"
/*
 * nd_noise.cu
 *
 *  Created on: 2018/04/21
 *      Author: kazuki
 */

#include <hiprand/hiprand_kernel.h>




__device__ static float noisegen(hiprandState_t * s, float mean = 0.0, float stddev =1.0)
{
	return stddev*hiprand_normal(s) + mean;
}

__device__ float3 getSystemNoise(hiprandState_t * s)
{

	struct mean_var
	{
		float x_mean = 0.0;
		float x_stddev = 0.0;
		float y_mean = 0.0;
		float y_stddev = 0.0;
		float theta_mean = 0.0;
		float theta_stddev = 0.0;
	};
	mean_var param;
	float3 v = { noisegen(s,param.x_mean,param.x_stddev), noisegen(s,param.y_mean,param.y_stddev), noisegen(s,param.theta_mean,param.theta_stddev) };
	return v;
}

__device__ float3 suggest_distribution(float3 x, float3 system_noise)
{
	//TODO:状態遷移関数?
	return x + system_noise;
}


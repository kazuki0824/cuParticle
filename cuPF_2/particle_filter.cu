#include "hip/hip_runtime.h"
/*
 * particle_filter.cu
 *
 *  Created on: 2018/10/11
 *      Author: maleicacid
 */

#include "devices/devices.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "modules/emicp.cuh"
#include "modules/cub_wrapper.cuh"
#include "particle_filter.h"
#include "user/likelihood.cuh"
#include "user/behavior.h"


#include <stdio.h>
static int b_search(float ary[], float key, int imin, int imax) {
	if (imax < imin) {
		return imax;
	} else {
		int imid = imin + (imax - imin) / 2;
		if (ary[imid] > key) {
			return b_search(ary, key, imin, imid - 1);
		} else if (ary[imid] < key) {
			return b_search(ary, key, imid + 1, imax);
		} else {
			return imid;
		}
	}
}
static inline int sampling(float random_seed, float ary[], int count)
{
	int imax = count - 1;
	int index = b_search(ary, random_seed, 0, imax);
	if (index>=imax) return imax;
	else return index + 1;
}


/*********************************************************/

__device__ float map[MAP_SIZE*MAP_SIZE];
static void SetupLMap(float * from, size_t count)
{
	hipMemcpyToSymbol(HIP_SYMBOL(map), from, count);
}

/*********************************************************/

// 位置情報
extern float3 state;

float * p;
float2 * dparticle;
float2 hparticle[sample_count];
float * dLikelihood_table;
float hLikelihood_table[sample_count];
static void prepare_particle_likelihood(float3 xy)
{
	// パーティクルの集合について、尤度と位置を別々に確保している
	hipMalloc((float2**)&dparticle,sample_count * sizeof(float2));
	hipMalloc((float2**)&dLikelihood_table,sample_count * sizeof(float));
	//TODO: hparticleに撒く・hLikelihood_tableに尤度をセット
	
	hipMemcpy(dparticle, hparticle, sizeof(float2) * sample_count,hipMemcpyDeviceToHost);
	hipMemcpy(dLikelihood_table, hLikelihood_table, sizeof(float2) * sample_count,hipMemcpyDeviceToHost);
}


hiprandGenerator_t g;
hipStream_t stream_1;
hipStream_t stream_2;
void Init(float x, float y)
{
	state = make_float3(x,y,0);

	//TODO: 尤度マップ転送
	SetupLMap(NULL, MAP_SIZE*MAP_SIZE);

	//Init RNG (Host)
	hiprandCreateGenerator(&g, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(g,clock());
	hipHostAlloc(&p, sample_count * sizeof(float), hipHostMallocMapped);

	//Generate random values for first re-sampling
	float* dp; hipHostGetDevicePointer(&dp, p, 0);
	hiprandGenerateUniform(g, dp, sample_count);

	//Initialize particles
	prepare_particle_likelihood(state);

	//LRF zero-copy
	hipHostAlloc(&hLRF, sample_count * sizeof(float), hipHostMallocWriteCombined);

	//Create a new stream
	hipStreamCreate(&stream_1);
	hipStreamCreate(&stream_2);
}

__global__ static void kStep(float2 * particle_device, float2 * LRF_device, float * LT_device,float3 x_y, unsigned int seed, float * map_device, int n_Beam)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t s;
	hiprand_init(seed, idx, 0, &s);

	particle_device[idx] = prediction(particle_device[idx], &s);
	LT_device[idx] = likelihood(LT_device[idx], particle_device[idx], LRF_device, n_Beam, map_device);
}
void Step()
{
	//Prediction update, likelihood(null stream)
	float2* dLRF; hipHostGetDevicePointer(&dLRF, hLRF, 0);
	kStep<<<64,128>>>(dparticle,dLRF,dLikelihood_table,state, clock(), map, nBeam);

	//Inclusive scan using CUB(null stream)
	float hPrefix[sample_count];
	incl_scan_CDF(dLikelihood_table, hPrefix, sample_count, 0);

	//TODO: ここか、main()の中でICPを行う？タイミングは任せる
	float2 hICP_result;

	//Wait
	hipStreamSynchronize(stream_1);
	//Copy ICP result(null stream/blocking)

	//Generate random values for "next" re-sampling
	float* dp; hipHostGetDevicePointer(&dp, p, 0);
	hiprandGenerateUniform(g, dp, sample_count);

	//Resampling (Binary search method)
	float max_l = hPrefix[sample_count-1];
	float2 new_particles[sample_count] = {0};
	float new_likelihood[sample_count] = {0};
	for (int var = 0; var < sample_count; ++var) {
		int particle_index = sampling(p[var] * max_l, hPrefix, sample_count);
		new_particles[var] = hparticle[particle_index];
		new_likelihood[var] = hLikelihood_table[particle_index];
	}
	//TODO:Update hLikelihood_table
/*
 *
 */


	//Send re-sampled particles to GPU
	hipMemcpy(hparticle, new_particles, sizeof(float2) * sample_count,hipMemcpyHostToHost);
	hipMemcpy(dparticle, hparticle, sizeof(float2) * sample_count,hipMemcpyDeviceToHost);
	hipMemcpy(dLikelihood_table, hLikelihood_table, sizeof(float2) * sample_count,hipMemcpyDeviceToHost);
}

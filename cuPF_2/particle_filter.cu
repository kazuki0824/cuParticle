#include "hip/hip_runtime.h"
/*
 * particle_filter.cu
 *
 *  Created on: 2018/10/11
 *      Author: maleicacid
 */


#include <cub/device/device_scan.cuh>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "particle_filter.h"
#include "user/likelihood.h"

#include <stdio.h>
int b_search(float ary[], float key, int imin, int imax) {
    if (imax < imin) {
        return imax;
    } else {
        int imid = imin + (imax - imin) / 2;
        if (ary[imid] > key) {
            return b_search(ary, key, imin, imid - 1);
        } else if (ary[imid] < key) {
            return b_search(ary, key, imid + 1, imax);
        } else {
            return imid;
        }
    }
}


/*********************************************************/

float3 state = {0};
static void SetupLMap() //尤度マップ転送
{

}


float * p;
hiprandGenerator_t g;
void Init(float x, float y, float z)
{
	state = make_float3(x,y,z);

	SetupLMap();

	//Init RNG (Host)
	hiprandCreateGenerator(&g, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(g,clock());

}
void Step(vParameter param)
{
	hipHostAlloc(&p, 8192 * sizeof(float), hipHostMallocMapped); //ゼロコピーメモリ

	//Generate random values for re-sampling
	float* dp; hipHostGetDevicePointer(&dp, p, 0);
	hiprandGenerateUniform(g, dp, sample_count);


}
